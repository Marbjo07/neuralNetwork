#include "hip/hip_runtime.h"
﻿#pragma once

#include "NeuralNetwork.cuh"

#ifndef NEURALNETWORK_CPP
#define NEURALNETWORK_CPP


//
// Layer class:
//


NeuralNet::Layer::ANN::ANN(int numberOfNeurons, int numberOfNeuronsPrevLayer, const float defualtWeight) {

    m_numberNeurons = numberOfNeurons;
    hipMalloc(&d_activations, sizeof(float) * numberOfNeurons);
    

    dim3 DimGrid(GRID_SIZE_NEURALNETWORK, GRID_SIZE_NEURALNETWORK, 1);
    dim3 DimBlock(BLOCK_SIZE_NEURALNETWORK, BLOCK_SIZE_NEURALNETWORK, 1);
    

    hipMalloc(&d_weights, sizeof(float) * numberOfNeurons * numberOfNeuronsPrevLayer);
    if (defualtWeight != NULL) {
        GpuHelperFunc::setAllValuesInArrayToOneVal << <DimBlock, DimGrid >> > (d_weights, numberOfNeurons * numberOfNeuronsPrevLayer, defualtWeight);
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::Layer::ANN::ANN()");
        
        hipDeviceSynchronize();
    }
    else {
        // Random numbers between -1 and 1
        Random::ArrayGpu << < DimBlock, DimGrid >> > (d_weights, numberOfNeurons * numberOfNeuronsPrevLayer, Random::offset + std::rand());
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::Layer::ANN::ANN()");
        
        hipDeviceSynchronize();
    }

}


void NeuralNet::Layer::ANN::setActivation(std::vector<float> a) {
    hipMemcpy(d_activations, &a[0], m_numberNeurons * sizeof(float), hipMemcpyHostToDevice);
}

float* NeuralNet::Layer::ANN::getActivations() {

    float* out;
    out = (float*)malloc(m_numberNeurons * sizeof(float));
    hipMemcpy(out, d_activations, m_numberNeurons * sizeof(float), hipMemcpyHostToDevice);
    return out;
}

//
// NeuralNet class:
//


void NeuralNet::addLayer(int numberOfNeurons) {
    m_shape.push_back(numberOfNeurons);
}

void NeuralNet::setInput(std::vector<float> input) {
    if (m_numberLayers <= 0) {
        std::cout << "\033[1;31 ERROR:\033[0m In setInput() no valid layers. Number of layers: " << m_numberLayers << " Caused by : " << m_name << std::endl;
        return;
    }
    if (input.size() != m_shape[0]) {
        std::cout << "\033[1;31 ERROR:\033[0m In setInput() input size not matching networks first layer make sure to call init(). Caused by: " << m_name << std::endl;
        return;
    }

    m_layers[0].setActivation(input);
}

void NeuralNet::setRandomInput() {
    if (m_numberLayers <= 0){
        std::cout << "\033[1;31ERROR:\033[0m In setRandomInput() no valid layers. Number of layers: " << m_numberLayers << " Caused by : " << m_name << std::endl;
        return;
    }
    dim3 DimGrid(GRID_SIZE_NEURALNETWORK, GRID_SIZE_NEURALNETWORK, 1);
    dim3 DimBlock(BLOCK_SIZE_NEURALNETWORK, BLOCK_SIZE_NEURALNETWORK, 1);
    
    Random::ArrayGpu << < DimBlock, DimGrid >> > (m_layers.front().d_activations, m_layers.front().m_numberNeurons, Random::offset + std::rand());

    CHECK_FOR_KERNEL_ERRORS("NeuralNet::setRandomInput()");

    hipDeviceSynchronize();


}

void NeuralNet::init(std::string name, const float defualtWeight) {

    m_name = name;

    if (m_totalNumberOfNeurons >= m_shape[0]) {
        for (auto i = 0; i < m_shape.size(); i++) {
            if (m_shape[i] != m_layers[i].m_numberNeurons) {
                m_layers.clear();
                break;
            }
        }
        this->random();
        return;
    }

    m_layers.reserve(m_shape.size());
    m_numberLayers = (uint32_t)m_shape.size();

    // Adds placeholder neurons
    m_layers.emplace_back(Layer::ANN(m_shape[0]));
    m_totalNumberOfNeurons = m_shape[0];

    for (int i = 1; i < m_shape.size() ; i++) {
        m_layers.emplace_back(Layer::ANN(m_shape[i], m_shape[i-1], defualtWeight));
        m_totalNumberOfNeurons += m_shape[i];
    }

}

/**
  This function cant accept \
 Please use /
 *
 */
void NeuralNet::save(std::string path) {

    std::cout << "Saving... \n";

    std::ofstream saveFile(path, std::ios_base::binary);

    if (saveFile.is_open()) {

        // Save name of model
        saveFile.write(m_name.c_str(), m_name.size());
        saveFile.write("\0", sizeof(char)); 

        // Save size of m_shape
        uint32_t sizeOfShape = (uint32_t)m_shape.size();
        saveFile.write(reinterpret_cast<const char*>(&sizeOfShape), sizeof(int));


        // Save shape of model
        saveFile.write(reinterpret_cast<const char*>(&m_shape[0]), m_shape.size() * sizeof(int));



        // Save weights
        for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {

            float* tmp;
            tmp = (float*)malloc(m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons * sizeof(float));
            hipMemcpy(tmp, m_layers[layerNum].d_weights, m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons * sizeof(float), hipMemcpyDeviceToHost);

            for (uint32_t weightNum = 0; weightNum < m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons; weightNum++) {
                saveFile.write((const char*)&tmp[weightNum], sizeof(float));
            }
            free(tmp);
        }

        // Save bias
        for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {
         
            saveFile.write((const char*)&m_layers[layerNum].m_bias, sizeof(float));

        }



        saveFile.close();
        std::cout << "Done saving!\n";

    }
    else {
        std::cout << "\033[1;31ERROR\033[0m in save() failed to save model. Caused by: " << m_name << std::endl;
    }


}

void NeuralNet::load(std::string path) {
    std::cout << "Loading pre trained model...\n";

    std::ifstream loadFile(path, std::ios_base::binary);

    if (loadFile.is_open()) {


        // Get name
        std::string modelName;
        
        std::getline(loadFile, modelName, '\0');

        int sizeOfShape;

        // Get number of layers
        loadFile.read((char*)&sizeOfShape, sizeof(sizeOfShape));
        m_shape.resize(sizeOfShape);
        
        // Get shape of model
        loadFile.read(reinterpret_cast<char*>(&m_shape[0]), sizeOfShape * sizeof(int));
        
        // Initialize without random weights
        init(modelName);


        // Load value of weights
        for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {
            
            float* tmp;
            tmp = (float*)malloc(m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons * sizeof(float));
            for (uint32_t weightNum = 0; weightNum < m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons; weightNum++) {

                loadFile.read((char*)&tmp[weightNum], sizeof(float));

            }
            hipMemcpy(m_layers[layerNum].d_weights, tmp, m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons * sizeof(float), hipMemcpyHostToDevice);


        }

        // Load value of bias
        for (uint32_t layer = 1; layer < m_numberLayers; layer++) {
            loadFile.read((char*)&m_layers[layer].m_bias, sizeof(float));
        }

        loadFile.close();
    }
    else {
        std::cout << "Could not open file\n";
    }

    std::cout << "\nDone loading pre trained model\n";
}



void NeuralNet::printWeightsAndBias() {

    std::cout << "Weights: \n";

    // every colum is the weights for one neuron
    for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {
        GpuHelperFunc::printArray<<<1, 1>>>(m_layers[layerNum].d_weights, m_layers[layerNum - 1].m_numberNeurons * m_layers[layerNum].m_numberNeurons);
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::printWeightsAndBias()");
        hipDeviceSynchronize();
    
        printf("\n");
    }


    std::cout << "Bias: \n";
    for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {
        printf(" %.6f ", m_layers[layerNum].m_bias);
    }


    std::cout << "\n\n\n";

}

void NeuralNet::printActivations() {

    printf("Activations: ");
    
    for (uint32_t layerNum = 0; layerNum < m_numberLayers; layerNum++) {
        GpuHelperFunc::printArray << <1, 1 >> > (m_layers[layerNum].d_activations, m_layers[layerNum].m_numberNeurons);
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::printActivations()");
        
        hipDeviceSynchronize();
    }

}

void NeuralNet::random() {
    
    dim3 DimGrid(GRID_SIZE_NEURALNETWORK, GRID_SIZE_NEURALNETWORK, 1);
    dim3 DimBlock(BLOCK_SIZE_NEURALNETWORK, BLOCK_SIZE_NEURALNETWORK, 1);
    
    for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {

        Random::ArrayGpu << < DimBlock, DimGrid >> > (m_layers[layerNum].d_weights, m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons, Random::offset + rand());
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::random()");
        
        hipDeviceSynchronize();

        m_layers[layerNum].m_bias = Random::Default();
    }
}

void NeuralNet::mutate(float mutationStrength) {
    
    dim3 DimGrid(GRID_SIZE_NEURALNETWORK, GRID_SIZE_NEURALNETWORK, 1);
    dim3 DimBlock(BLOCK_SIZE_NEURALNETWORK, BLOCK_SIZE_NEURALNETWORK, 1);

    for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {

        Random::MutateArrayGpu << < DimBlock, DimGrid >> > (m_layers[layerNum].d_weights, m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons, Random::offset + rand());

        CHECK_FOR_KERNEL_ERRORS("NeuralNet::mutate()");

        hipDeviceSynchronize();

        m_layers[layerNum].m_bias *= Random::Default();
    }
}


float NeuralNet::sumOfWeightsAndBias() {
    float sum = 0;

    for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {

        GpuHelperFunc::sumOfArray<<<1, 1>>>(m_layers[layerNum].d_weights, m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons, sum);
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::sumOfWeightsAndBias()");

        sum += m_layers[layerNum].m_bias;
    }
    return sum;
}

float* NeuralNet::getOutput() {

    float* output;
    output = (float*)malloc(m_layers.back().m_numberNeurons * sizeof(float));
    hipMemcpy(output, m_layers.back().d_activations, m_layers.back().m_numberNeurons * sizeof(float), hipMemcpyDeviceToHost);

    return output;
}

float NeuralNet::MAELossFunction(float* output, std::vector<float> target) {
    float error = 0; 
    
    for (size_t i = 0; i < target.size(); i++) {
        error += std::abs(output[i] - target[i]);
    }

    return std::abs(error) / target.size();

}

float NeuralNet::MSELossFunction(float* output, std::vector<float> target){
    float error = 0;

    for (size_t i = 0; i < target.size(); i++) {
        error += std::powf(output[i] - target[i], 2);
    }

    return std::abs(error) / target.size();
}

float NeuralNet::LossFunction(float* output, std::vector<float> target) {

    return MAELossFunction(output, target);

}

float NeuralNet::performTest(std::vector<std::vector<float>> testData, std::vector<std::vector<float>> expectedOutput ) {

    float error = 0;

    for (size_t i = 0; i < testData.size(); i++) {

        setInput(testData[i]);
        
        error += LossFunction(feedForward(), expectedOutput[i]);
    }

    return error;
}

void NeuralNet::printOutput() {

    std::vector<float> output;
    output.resize(m_shape.back());

    memcpy(&output[0], getOutput(), m_shape.back() * sizeof(float));


    printf("Output: ");
    for (auto i = 0; i < output.size(); i++) std::cout << output[i] << " ";
    printf("\n");
}

void NeuralNet::optimizeGridsAndBlocksFeedforward(uint32_t maxGrid, uint32_t maxBlock, uint32_t numberOfTest) {

    uint32_t bestGrid = 1;
    uint32_t bestBlock = 1;


    auto start = std::chrono::high_resolution_clock::now();
    feedForward(1, 1);
    auto minDuration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start).count();

    for (uint32_t grid = 1; grid <= maxGrid; ++grid) {
        for (uint32_t block = 2; block <= maxBlock; ++block) {

            auto start = std::chrono::high_resolution_clock::now();
            for (auto k = 0; k < numberOfTest; ++k) {
                feedForward(grid, block);
            }
            auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now() - start).count();

            if (duration < minDuration) {
                minDuration = duration;
                bestBlock = block;
                bestGrid = grid;
            }

        }
    }

    m_gridFeedforward = bestGrid;
    m_blockFeedforward = bestBlock;

    printf("Optimized Grid and Block for feedforward: (%d, %d)\n", bestGrid, bestBlock);



    
}

void NeuralNet::printSize() {
    uint64_t numberOfVaribles = 0;

    // Activation
    for (auto x : m_shape) {
        numberOfVaribles += x;
    }

    // Bias
    numberOfVaribles += m_shape.size();

    // Weights
    for (uint32_t i = 1; i < m_numberLayers; i++) {
        numberOfVaribles += m_shape[i - 1] * m_shape[i];
    }

    std::cout << "Number of parameter in neuralNetwork: " << numberOfVaribles
        << " size in bytes: " << numberOfVaribles * sizeof(float) << "\n"
        << " size in MB: " << (float)numberOfVaribles * sizeof(float) / 1024 / 1024 << "\n"
        << " size in GB: " << (float)numberOfVaribles * sizeof(float) / 1024 / 1024 / 1024 << "\n";
}

#endif // !NEURALNETWORK_CPP