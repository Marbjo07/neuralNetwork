#include "hip/hip_runtime.h"
﻿#pragma once

#include "NeuralNetwork.cuh"

#ifndef NEURALNETWORK_CPP
#define NEURALNETWORK_CPP


//
// Layer class:
//


NeuralNet::Layer::ANN::ANN(int numberOfNeurons, int numberOfNeuronsPrevLayer, const float defualtWeight) {

    m_numberNeurons = numberOfNeurons;
    hipMalloc(&d_activations, sizeof(float) * numberOfNeurons);
    

    dim3 DimGrid(GRID_SIZE_NEURALNETWORK, GRID_SIZE_NEURALNETWORK, 1);
    dim3 DimBlock(BLOCK_SIZE_NEURALNETWORK, BLOCK_SIZE_NEURALNETWORK, 1);
    

    hipMalloc(&d_weights, sizeof(float) * numberOfNeurons * numberOfNeuronsPrevLayer);
    if (defualtWeight != NULL) {
        GpuHelperFunc::setAllValuesInArrayToOneVal << <DimBlock, DimGrid >> > (d_weights, numberOfNeurons * numberOfNeuronsPrevLayer, defualtWeight);
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::Layer::ANN::ANN()");
        
        hipDeviceSynchronize();
    }
    else {
        // Random numbers between -1 and 1
        Random::ArrayGpu << < DimBlock, DimGrid >> > (d_weights, numberOfNeurons * numberOfNeuronsPrevLayer, Random::offset + std::rand());
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::Layer::ANN::ANN()");
        
        hipDeviceSynchronize();
    }

}


void NeuralNet::Layer::ANN::setActivation(std::vector<float> a) {
    hipMemcpy(d_activations, &a[0], m_numberNeurons * sizeof(float), hipMemcpyHostToDevice);
}

float* NeuralNet::Layer::ANN::getActivations() {

    float* out;
    out = (float*)malloc(m_numberNeurons * sizeof(float));
    hipMemcpy(out, d_activations, m_numberNeurons * sizeof(float), hipMemcpyHostToDevice);
    return out;
}

//
// NeuralNet class:
//


void NeuralNet::addLayer(int numberOfNeurons) {
    m_shape.push_back(numberOfNeurons);
}

void NeuralNet::setInput(std::vector<float> input) {
    if (m_numberLayers <= 0) {
        std::cout << "\033[1;31 ERROR:\033[0m In setInput() no valid layers. Number of layers: " << m_numberLayers << " Caused by : " << m_name << std::endl;
        return;
    }
    if (input.size() != m_shape[0]) {
        std::cout << "\033[1;31 ERROR:\033[0m In setInput() input size not matching networks first layer make sure to call init(). Caused by: " << m_name << std::endl;
        return;
    }

    m_layers[0].setActivation(input);
}

void NeuralNet::setRandomInput() {
    if (m_numberLayers <= 0){
        std::cout << "\033[1;31ERROR:\033[0m In setRandomInput() no valid layers. Number of layers: " << m_numberLayers << " Caused by : " << m_name << std::endl;
        return;
    }
    dim3 DimGrid(GRID_SIZE_NEURALNETWORK, GRID_SIZE_NEURALNETWORK, 1);
    dim3 DimBlock(BLOCK_SIZE_NEURALNETWORK, BLOCK_SIZE_NEURALNETWORK, 1);
    
    Random::ArrayGpu << < DimBlock, DimGrid >> > (m_layers.front().d_activations, m_layers.front().m_numberNeurons, Random::offset + std::rand());

    CHECK_FOR_KERNEL_ERRORS("NeuralNet::setRandomInput()");

    hipDeviceSynchronize();


}

void NeuralNet::init(std::string name, const float defualtWeight) {

    m_name = name;

    if (m_totalNumberOfNeurons >= m_shape[0]) {
        for (auto i = 0; i < m_shape.size(); i++) {
            if (m_shape[i] != m_layers[i].m_numberNeurons) {
                m_layers.clear();
                break;
            }
        }
        this->random();
        return;
    }

    m_layers.reserve(m_shape.size());
    m_numberLayers = (uint32_t)m_shape.size();

    // Adds placeholder neurons
    m_layers.emplace_back(Layer::ANN(m_shape[0]));
    m_totalNumberOfNeurons = m_shape[0];

    for (int i = 1; i < m_shape.size() ; i++) {
        m_layers.emplace_back(Layer::ANN(m_shape[i], m_shape[i-1], defualtWeight));
        m_totalNumberOfNeurons += m_shape[i];
    }

}

/**
  This function cant accept \
 Please use /
 *
 */
void NeuralNet::save(std::string path) {

    std::cout << "Saving... \n";

    std::ofstream saveFile(path, std::ios_base::binary);

    if (saveFile.is_open()) {

        // Save name of model
        saveFile.write(m_name.c_str(), m_name.size());
        saveFile.write("\0", sizeof(char)); 

        // Save size of m_shape
        uint32_t sizeOfShape = (uint32_t)m_shape.size();
        saveFile.write(reinterpret_cast<const char*>(&sizeOfShape), sizeof(int));


        // Save shape of model
        saveFile.write(reinterpret_cast<const char*>(&m_shape[0]), m_shape.size() * sizeof(int));



        // Save weights
        for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {

            float* tmp;
            tmp = (float*)malloc(m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons * sizeof(float));
            hipMemcpy(tmp, m_layers[layerNum].d_weights, m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons * sizeof(float), hipMemcpyDeviceToHost);

            for (uint32_t weightNum = 0; weightNum < m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons; weightNum++) {
                saveFile.write((const char*)&tmp[weightNum], sizeof(float));
            }
            free(tmp);
        }

        // Save bias
        for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {
         
            saveFile.write((const char*)&m_layers[layerNum].d_bias, sizeof(float));

        }



        saveFile.close();
        std::cout << "Done saving!\n";

    }
    else {
        std::cout << "\033[1;31ERROR\033[0m in save() failed to save model. Caused by: " << m_name << std::endl;
    }


}

void NeuralNet::load(std::string path) {
    std::cout << "Loading pre trained model...\n";

    std::ifstream loadFile(path, std::ios_base::binary);

    if (loadFile.is_open()) {


        // Get name
        std::string modelName;
        
        std::getline(loadFile, modelName, '\0');

        int sizeOfShape;

        // Get number of layers
        loadFile.read((char*)&sizeOfShape, sizeof(sizeOfShape));
        std::cout << "Sizeof shape: " << sizeOfShape << std::endl;
        m_shape.resize(sizeOfShape);
        
        // Get shape of model
        loadFile.read(reinterpret_cast<char*>(&m_shape[0]), sizeOfShape * sizeof(int));
        
        // Initialize without random weights
        init(modelName);


        // Load value of weights
        for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {
            
            float* tmp;
            tmp = (float*)malloc(m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons * sizeof(float));
            for (uint32_t weightNum = 0; weightNum < m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons; weightNum++) {

                loadFile.read((char*)&tmp[weightNum], sizeof(float));

            }
            hipMemcpy(m_layers[layerNum].d_weights, tmp, m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons * sizeof(float), hipMemcpyHostToDevice);


        }

        // Load value of bias
        for (uint32_t layer = 1; layer < m_numberLayers; layer++) {
            loadFile.read((char*)&m_layers[layer].d_bias, sizeof(float));
        }

        loadFile.close();
    }
    else {
        std::cout << "Could not open file\n";
    }

    std::cout << "\nDone loading pre trained model\n";
}



void NeuralNet::printWeightsAndBias() {

    std::cout << "Weights: \n";

    // every colum is the weights for one neuron
    for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {
        GpuHelperFunc::printArray<<<1, 1>>>(m_layers[layerNum].d_weights, m_layers[layerNum - 1].m_numberNeurons * m_layers[layerNum].m_numberNeurons);
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::printWeightsAndBias()");
        hipDeviceSynchronize();
    
        printf("\n");
    }


    std::cout << "Bias: \n";
    for (uint32_t layerNum = 0; layerNum < m_numberLayers; layerNum++) {
        printf(" %.6f ", m_layers[layerNum].d_bias);
    }


    std::cout << "\n\n\n";

}

void NeuralNet::printActivations() {

    std::cout << "Activations: " << std::endl;
    printf("n: %d", m_numberLayers);
    for (uint32_t layerNum = 0; layerNum < m_numberLayers; layerNum++) {
        GpuHelperFunc::printArray << <1, 1 >> > (m_layers[layerNum].d_activations, m_layers[layerNum].m_numberNeurons);
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::printActivations()");
        
        hipDeviceSynchronize();
    }

}

void NeuralNet::random() {
    
    dim3 DimGrid(GRID_SIZE_NEURALNETWORK, GRID_SIZE_NEURALNETWORK, 1);
    dim3 DimBlock(BLOCK_SIZE_NEURALNETWORK, BLOCK_SIZE_NEURALNETWORK, 1);
    
    for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {

        Random::ArrayGpu << < DimBlock, DimGrid >> > (m_layers[layerNum].d_weights, m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons, Random::offset + rand());
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::random()");
        
        hipDeviceSynchronize();

        m_layers[layerNum].d_bias = Random::Default();
    }
}

void NeuralNet::mutate(float mutationStrength) {
    
    dim3 DimGrid(GRID_SIZE_NEURALNETWORK, GRID_SIZE_NEURALNETWORK, 1);
    dim3 DimBlock(BLOCK_SIZE_NEURALNETWORK, BLOCK_SIZE_NEURALNETWORK, 1);

    for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {

        Random::MutateArrayGpu << < DimBlock, DimGrid >> > (m_layers[layerNum].d_weights, m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons, Random::offset + rand());

        CHECK_FOR_KERNEL_ERRORS("NeuralNet::mutate()");

        hipDeviceSynchronize();

        m_layers[layerNum].d_bias *= Random::Default();
    }
}


float NeuralNet::sumOfWeightsAndBias() {
    float sum = 0;

    for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {

        GpuHelperFunc::sumOfArray<<<1, 1>>>(m_layers[layerNum].d_weights, m_layers[layerNum].m_numberNeurons * m_layers[layerNum - 1].m_numberNeurons, sum);
        
        CHECK_FOR_KERNEL_ERRORS("NeuralNet::sumOfWeightsAndBias()");

        sum += m_layers[layerNum].d_bias;
    }
    return sum;
}

float* NeuralNet::getOutput() {

    float* output;
    output = (float*)malloc(m_layers.back().m_numberNeurons * sizeof(float));
    hipMemcpy(output, m_layers.back().d_activations, m_layers.back().m_numberNeurons * sizeof(float), hipMemcpyDeviceToHost);

    return output;
}

float NeuralNet::MAELossFunction(float* output, std::vector<float> target) {
    float error = 0; 
    
    for (size_t i = 0; i < target.size(); i++) {
        error += std::abs(output[i] - target[i]);
    }

    return std::abs(error) / target.size();

}

float NeuralNet::MSELossFunction(float* output, std::vector<float> target){
    float error = 0;

    for (size_t i = 0; i < target.size(); i++) {
        error += std::powf(output[i] - target[i], 2);
    }

    return std::abs(error) / target.size();
}

float NeuralNet::LossFunction(float* output, std::vector<float> target) {

    return MAELossFunction(output, target);

}

float NeuralNet::performTest(std::vector<std::vector<float>> testData, std::vector<std::vector<float>> expectedOutput ) {

    float error = 0;

    for (size_t i = 0; i < testData.size(); i++) {

        setInput(testData[i]);
        
        error += LossFunction(feedForward(), expectedOutput[i]);
    }

    return error;
}

void NeuralNet::printOutput() {

    std::vector<float> output;
    output.reserve(m_shape.back() * sizeof(float));

    memcpy(&output[0], getOutput(), m_shape.back() * sizeof(float));


    printf("Output: ");
    for (auto i = 0; i < output.size(); i++) std::cout << output[i] << " | ";
    printf("\n");
}

#endif // !NEURALNETWORK_CPP