#include "hip/hip_runtime.h"
#pragma once

#include "NeuralNetwork.hpp"
#include "E:\CUDA\Cuda Development\include\hip/hip_runtime.h"
#include "E:\CUDA\Cuda Development\include\hip/hip_runtime.h"
#include "E:\CUDA\Cuda Development\include\"

#ifndef FEEDFORWARD_CPP
#define FEEDFORWARD_CPP

__global__ void multiply(float* activations, float* weights, int* size, float* writeVal) {

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    while (tid < *size) {
        (*writeVal) += activations[tid] + weights[tid];    
        tid += blockDim.x;      
    }
}


#define STEPSIZE 8
std::vector<float> NeuralNet::feedForward() {


    uint32_t neuronNum;

    for (uint32_t layerNum = 1; layerNum < m_numberLayers; layerNum++) {
        float* prevLayerActivation;

        hipMalloc(&prevLayerActivation, m_layers[layerNum - 1].m_numberNeurons * sizeof(float));

        hipMemcpy(prevLayerActivation, m_layers[layerNum].getActivation().data(), m_layers[layerNum - 1].m_numberNeurons * sizeof(float), hipMemcpyHostToDevice);

        for (neuronNum = 0; neuronNum < m_layers[layerNum].m_numberNeurons; neuronNum++) {
            
            float* weights;


            hipMalloc(&weights, m_layers[layerNum - 1].m_numberNeurons * sizeof(float));

            hipMemcpy(weights, m_layers[layerNum].m_neurons[neuronNum].m_weights.data(), m_layers[layerNum - 1].m_numberNeurons * sizeof(float), hipMemcpyHostToDevice);



            multiply<<<2,256>>>(prevLayerActivation, weights, &m_layers[layerNum - 1].m_numberNeurons, &m_layers[layerNum].m_neurons[neuronNum].m_activation);

            hipDeviceSynchronize();
            hipFree(weights);
        }

        hipFree(prevLayerActivation);

        if (STEPSIZE < m_layers[layerNum].m_numberNeurons) {
            for (neuronNum = 0; neuronNum < m_layers[layerNum].m_numberNeurons - STEPSIZE; neuronNum += STEPSIZE) {

                m_layers[layerNum].m_neurons[neuronNum + 0].m_activation = m_layers[layerNum].m_neurons[neuronNum + 0].activationFunction(m_layers[layerNum].m_neurons[neuronNum + 0].m_activation);
                m_layers[layerNum].m_neurons[neuronNum + 1].m_activation = m_layers[layerNum].m_neurons[neuronNum + 1].activationFunction(m_layers[layerNum].m_neurons[neuronNum + 1].m_activation);
                m_layers[layerNum].m_neurons[neuronNum + 2].m_activation = m_layers[layerNum].m_neurons[neuronNum + 2].activationFunction(m_layers[layerNum].m_neurons[neuronNum + 2].m_activation);
                m_layers[layerNum].m_neurons[neuronNum + 3].m_activation = m_layers[layerNum].m_neurons[neuronNum + 3].activationFunction(m_layers[layerNum].m_neurons[neuronNum + 3].m_activation);
                m_layers[layerNum].m_neurons[neuronNum + 4].m_activation = m_layers[layerNum].m_neurons[neuronNum + 4].activationFunction(m_layers[layerNum].m_neurons[neuronNum + 4].m_activation);
                m_layers[layerNum].m_neurons[neuronNum + 5].m_activation = m_layers[layerNum].m_neurons[neuronNum + 5].activationFunction(m_layers[layerNum].m_neurons[neuronNum + 5].m_activation);
                m_layers[layerNum].m_neurons[neuronNum + 6].m_activation = m_layers[layerNum].m_neurons[neuronNum + 6].activationFunction(m_layers[layerNum].m_neurons[neuronNum + 6].m_activation);
                m_layers[layerNum].m_neurons[neuronNum + 7].m_activation = m_layers[layerNum].m_neurons[neuronNum + 7].activationFunction(m_layers[layerNum].m_neurons[neuronNum + 7].m_activation);
            }
        }
        for (; neuronNum < m_layers[layerNum].m_numberNeurons; neuronNum++) {

            m_layers[layerNum].m_neurons[neuronNum].m_activation = m_layers[layerNum].m_neurons[neuronNum].activationFunction(m_layers[layerNum].m_neurons[neuronNum].m_activation);

        }
    }

    return m_layers.back().getActivation();
}
#undef STEPSIZE

#endif // !FEEDFORWARD_CPP